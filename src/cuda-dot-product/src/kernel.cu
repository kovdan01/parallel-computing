#include "hip/hip_runtime.h"
#include <interfaces.hpp>

#include <cuda/runtime_api.hpp>

namespace my::cuda
{

namespace
{
    
template <typename T>
__global__ void multiply_kernel(const T* __restrict__ a,
                                const T* __restrict__ b,
                                T* __restrict__ c,
                                int size)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < size)
        c[tid] = a[tid] * b[tid];
}

}  // namespace

void multiply(const double* a,
              const double* b,
              double* c,
              int size,
              int blocks_per_grid,
              int threads_per_block)
{
    ::cuda::launch(multiply_kernel<double>,
                   ::cuda::launch_configuration_t(blocks_per_grid, threads_per_block),
                   a, b, c, size);
}

namespace
{

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template <class T>
struct SharedMemory
{
    __device__ inline operator T*()
    {
        extern __shared__ int __smem[];
        return reinterpret_cast<T*>(__smem);
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return reinterpret_cast<T*>(__smem);
    }
};

// Specialize for double to avoid unaligned memory
// access compile errors
template <>
struct SharedMemory<double>
{
    __device__ inline operator double*()
    {
        extern __shared__ double __smem_d[];
        return reinterpret_cast<double*>(__smem_d);
    }

    __device__ inline operator const double*() const
    {
        extern __shared__ double __smem_d[];
        return reinterpret_cast<double*>(__smem_d);
    }
};

namespace cg = cooperative_groups;

template <typename T>
__global__ void reduce_kernel(const T* __restrict__ g_idata,
                              T* __restrict__ g_odata,
                              unsigned n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T* sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    T my_sum = (i < n) ? g_idata[i] : 0;

    if (i + blockDim.x < n)
    {
        my_sum += g_idata[i + blockDim.x];
    }

    sdata[tid] = my_sum;
    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] = my_sum = my_sum + sdata[tid + s];
        }
        cg::sync(cta);
    }

    // write result for this block to global mem
    if (tid == 0)
    {
        g_odata[blockIdx.x] = my_sum;
    }
}

}  // namespace

void reduce(const double* idata,
            double* odata,
            int size,
            int blocks_per_grid,
            int threads_per_block)
{
    dim3 dim_block(threads_per_block, 1, 1);
    dim3 dim_grid(blocks_per_grid, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smem_size = (threads_per_block <= 32)
                     ? 2 * threads_per_block * sizeof(double)
                     : threads_per_block * sizeof(double);

    ::cuda::launch(reduce_kernel<double>,
                   ::cuda::launch_configuration_t(dim_grid, dim_block, smem_size),
                   idata, odata, size);
}

}  // namespace my::cuda
